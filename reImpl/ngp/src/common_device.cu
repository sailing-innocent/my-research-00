#include "hip/hip_runtime.h"
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/tinyexr_wrapper.h>

#include <unsupported/Eigen/MatrixFunctions>

#include <stb_image/stb_image.h>

using namespace Eigen;
using namespace tcnn;

NGP_NAMESPACE_BEGIN

Matrix<float, 3, 4> log_space_lerp(const Matrix<float, 3, 4>& begin, const Matrix<float, 3, 4>& end, float t) {
    Matrix4f A = Matrix4f::Identity();
    A.block<3,4>(0,0) = begin;
    Matrix4f B = Matrix4f::Identity();
    B.block<3,4>(0,0) = end;

    Matrix4f log_space_a_to_b = (B * A.inverse()).log();

    return ((log_space_a_to_b * t).exp() * A).block<3,4>(0,0);
}

GPUMemory<float> laod_stbi(const std::string& filename, int& width, int& height) {
    bool is_hdr = stbi_is_hdr(filename.c_str());

    void* data; // width * height * RGBA
    int comp;

    if (is_hdr) {
        data = stbi_loadf(filename.c_str(), &width, &height, &comp, 4);
    } else {
        data = stbi_load(filename.c_str(), &width, &height, &comp, 4);
    }

    if (!data) {
        throw std::runtime_error{std::string{stbi_failure_reason()}};
    }

    ScopeGuard mem_guard{[&](){ stbi_image_free(data); }};

    if (width == 0 || height == 0) {
        throw std::runtime_error{"Image has zero pixels."};
    }

    GPUMemory<float> result(width * height * 4);
    if (is_hdr) {
        result.copy_from_host((float*)data);
    } else {
        GPUMemory<uint8_t> bytes(width * height * 4);
        bytes.copy_from_host((uint8*)data);
        // TODO: What is linear kernel?
        // linear_kernel(from_rgba32<float>, 0, nullptr, width * height, bytes.data(), false, false, 0);
    }
}

NGP_NAMESPACE_END