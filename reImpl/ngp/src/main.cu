#include "hip/hip_runtime.h"
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/testbed.h>

#include <tiny-cuda-nn/common.h>

// #include <args/args.hxx>
// #include <filesystem/path.h>

// using namespace args;
using namespace ngp;
using namespace std;
using namespace tcnn;

// namespace fs = ::filesystem;

int main(int argc, char** argv) {
    // parsers
    try {
        // mode
        ETestbedMode mode;
        // switch mode flag 
        mode = ETestbedMode::Nerf;

        // testbed
        Testbed testbed{mode};
        // testbed.load_traning_data()

        std::string mode_str;
        // switch mode str
        mode_str = "nerf";

        // snapshot?

        // gui?
        bool gui = true;
#ifndef NGP_GUI
    gui = false
#endif
        if (gui) {
            testbed.init_window(1920, 1080, false, false);
        }

        // Render/Training Loop
        //  while (testbed.frame()) {
            // if no gui, log
        // }
    } catch (const exception& e) {
        // error log
        return 1;
    }
}