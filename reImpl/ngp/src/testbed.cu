#include "hip/hip_runtime.h"

// project
#include <neural-graphics-primitives/common.h>
// #include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/testbed.h>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/loss.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/optimizer.h>
#include <tiny-cuda-nn/trainer.h>


#ifdef NGP_GUI
#   include <imgui/imgui.h>
#   include <imgui/backends/imgui_impl_glfw.h>
#   include <imgui/backends/imgui_impl_opengl3.h>
#   include <imguizmo/ImGuizmo.h>
#   include <stb_image/stb_image.h>
#   ifdef _WIN32
#       include <GL/gl3w.h>
#   else
#       include <GL/glew.h>
#   endif
#   include <GLFW/glfw3.h>
#endif

// Windows.h is evil, hahahaha
#undef min
#undef max
#undef near
#undef far

using namespace Eigen;
using namespace std::literals::chrono_literals;
using namespace tcnn;

NGP_NAMESPACE_BEGIN

bool Testbed::frame() {
#ifdef NGP_GUI
    // init window
    if (m_render_window) {
        if (!begin_frame_and_handle_user_input()) {
            return false;
        }
    }
#endif

    // Render against the trained neural network
    // we can skip rendering if the scene camera doesn't change
    // uint32_t n_to_skip = m_train ? tcnn::clamp(m_training_step / 16u, 15u, 255u) : 0;
    /*
    try {
        while (true) {
            // (*m_task_queue.tryPop())();
        }
    } catch (SharedQueueEmptyException&) {}
    */
    // train and render

#ifdef NGP_GUI 
    if (m_render_window) {
        if (m_gui_redraw) {
            // gather gui
        }
        // draw_gui()
        m_gui_redraw = false;
        m_last_gui_draw_time_point = std::chrono::steady_clock::now();
    }

    // ImGui::EndFrame
#endif 
    return true;
}

bool Testbed::begin_frame_and_handle_user_input() {
    bool ESCAPE_STATE = false;
    if (glfwWindowShouldClose(m_glfw_window) || ESCAPE_STATE) {
        destroy_window();
        return false;
    }

    {
        // update time
        auto now = std::chrono::steady_clock::now();
        auto elapsed = now - m_last_frame_time_point;
        m_last_frame_time_point = now;
        m_frame_ms.update(std::chrono::duration<float, std::milli>(elapsed).count());
    }

    glfwPollEvents();
    glfwGetFramebufferSize(m_glfw_window, &m_window_res.x(), &m_window_res.y());

    // IMGUI

    // NeRF special process

    // Keyboard Events
    // mouse_drag

    return true;
}

// merge_parent_network_config
// ends with
// load trining data

Testbed::Testbed(ETestbedMode mode)
: m_testbed_mode(mode)
{
    std::cout << "Testbed Init: " << std::endl;
}

Testbed::~Testbed() {
    // Destroy Window
}

void Testbed::init_window(int resw, int resh, bool hidden, bool second_window)
{
#ifndef NGP_GUI
	throw std::runtime_error{"init_window failed: NGP was built without GUI support"};
#else // define NGP_GUI
    std::cout << "Init Window" << std::endl; 
    // if not def NGP_GUI throw error
    m_window_res = { resw, resh };
    // glfw set error callback
    glfwSetErrorCallback(glfw_error_callback);
    // glfw init
    if (!glfwInit()) {
        throw std::runtime_error{"GLFW cound not be initialized."};
    }
// IF VULKAN SPECIAL
// ENDIF VULKAN

    // init GLFW Window Parameters
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GLFW_TRUE);
    glfwWindowHint(GLFW_VISIBLE, hidden ? GLFW_FALSE : GLFW_TRUE);
    std::string title = "SIRI: Neural Graphics Primitives (";
    switch (m_testbed_mode) {
        case ETestbedMode::Image: title += "Image"; break;
        case ETestbedMode::Nerf: title += "NeRf"; break;
        case ETestbedMode::Sdf: title += "SDF"; break;
        case ETestbedMode::Volume: title += "Volume"; break;
        default: title += "NOTITLE" ; break;
    }
    title += ")";
    // Create Window
    m_glfw_window = glfwCreateWindow(m_window_res.x(), m_window_res.y(), title.c_str(), NULL, NULL, NULL);
    if (m_glfw_window == NULL) {
        throw std::runtime_error("GLFW window could not be created");
    }
    glfwMakeContextCurrent(m_glfw_window);
#ifdef _WIN32
    if (gl3wInit()) {
        throw std::runtime_error("GL3W could not be initialized");
    }
#endif // _WIN32
    // OTHER DEBUGGING

    std::cout << "init window: " << title << std::endl;

    glfwSwapInterval(0); // Disable vsync
    // Set Window User Pointer
    glfwSetWindowUserPointer(m_glfw_window, this);
    
    // Set Drop Callback
    glfwSetDropCallback(m_glfw_window, [](GLFWwindow* window, int count, const char** path) {
        Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
        if (!testbed) {
            return;
        }
        //TODO testbed->redraw_gui_next_frame();
        for (int i = 0; i < count; i++) {
            // testbed->handle_file(paths[i])
        }
    })

    // Set Key Callback
    // Set Current PosCallback
    // Set ScrollCallback
    // Set WindowSize Callback
    glfwSetWindowSizeCallback(m_glfw_window, [](GLFWwindow* window, int width, int height) {
        Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
        if (testbed) {
            testbed->redraw_next_frame();
        }
    })
    // Set framebufferSizeCallback
    glfwSetFramebufferSizeCallback(m_glfw_window, [](GLFWwindow* window, int width, int height) {
        Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
        if (testbed) {
            testbed->redraw_next_frame();
        }
    })

    
    // scale seems no longer valid
    // float xscale, yscale;
    // glfwGetWindowContentScale(m_glfw_window, &xscale, &yscale);

    // IMGUI init

    // render texture
    //TODO: waiting for shared_queue and render_buffer 
    /*
    m_render_textures = { std::make_shared<GLTexture>() };
    m_render_surfaces.clear();
    m_render_surfaces.emplace_back(m_render_textures.front());
    m_pip_render_texture = std::make_shared<GLTexture>();
    m_pip_render_surface = std::make_unique<CudaRenderBuffer>(m_pip_render_texture);
    */
    // set render window 
    m_render_window = true;
    // second window
#endif // NGP_GUI
}

void Testbed::destroy_window() {
#ifndef NGP_GUI
    throw std::runtime_error("destroy window failed: NGP was built without GUI");
#else
    if (!m_render_window) {
        throw std::runtime_error("Window must be initialized before to be destroyed");
    }

    // m_render_surfaces.clear();
    // m_render_textures.clear();

    // m_pip_render_surface
    // m_pip_render_texture

    // dlss

    // IMGUI

    m_glfw_window = nullptr;
    m_render_window = false;
#endif // NGP_GUI
}




void glfw_error_callback(int error, const char* description) {
    // tlog::error 
}

NGP_NAMESPACE_END