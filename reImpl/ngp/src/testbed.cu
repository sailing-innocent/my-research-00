#include "hip/hip_runtime.h"

// project
#include <neural-graphics-primitives/common.h>
// #include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/testbed.h>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/loss.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/optimizer.h>
#include <tiny-cuda-nn/trainer.h>


#ifdef NGP_GUI
#   include <imgui/imgui.h>
#   include <imgui/backends/imgui_impl_glfw.h>
#   include <imgui/backends/imgui_impl_opengl3.h>
#   include <imguizmo/ImGuizmo.h>
#   ifdef _WIN32
#       include <GL/gl3w.h>
#   else
#       include <GL/glew.h>
#   endif
#   include <GLFW/glfw3.h>
#endif

// Windows.h is evil, hahahaha
#undef min
#undef max
#undef near
#undef far

using namespace Eigen;
using namespace std::literals::chrono_literals;
using namespace tcnn;

NGP_NAMESPACE_BEGIN

Testbed::Testbed(ETestbedMode mode)
: m_testbed_mode(mode)
{
    // config
}

Testbed::~Testbed() {
    // Destroy Window
}

void Testbed::init_window(int resw, int resh, bool hidden, bool second_window)
{
    // if not def NGP_GUI throw error
    m_window_res = { resw, resh };
    // glfw set error callback
    glfwSetErrorCallback(glfw_error_callback);
    // glfw init
    if (!glfwInit()) {
        throw std::runtime_error{"GLFW cound not be initialized."};
    }
// IF VULKAN SPECIAL
// ENDIF VULKAN

    // init GLFW Window Parameters
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GLFW_TRUE);
    glfwWindowHint(GLFW_VISIBLE, hidden ? GLFW_FALSE : GLFW_TRUE);
    std::string title = "SIRI: Neural Graphics Primitives (";
    switch (m_testbed_mode) {
        case ETestbedMode::Image: title += "Image"; break;
    }
    title += ")";
    
    // Create Window
    // m_glfw_window = glfwCreateWindow(m_window_res.x(), m_window_res.y(), title.c_str(), NULL, NULL);
    if (m_glfw_window == NULL) {
        throw std::runtime_error("GLFW window could not be created");
    }
    // OTHER DEBUGGING

    std::cout << "init window: " << title << std::endl;

    glfwSwapInterval(0); // Disable vsync

}

void glfw_error_callback(int error, const char* description) {
    // tlog::error 
}

NGP_NAMESPACE_END