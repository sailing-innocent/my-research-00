#include "hip/hip_runtime.h"
/**
 * @file testbed_image.cu
 * @author your name (you@domain.com)
 * @brief 
 * @version 0.1
 * @date 2022-10-02
 * 
 * @copyright Copyright (c) 2022
 * 
 */

#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/random_val.cuh>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/testbed.h>

#include <tiny-cuda-nn/gpu_matrix.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/network.h>
#include <tniy-cuda-nn/trainer.h>

#include <fstream>

using namespace Eigen;
using namespace tcnn;

NGP_NAMESPACE_BEGIN

// network_dims_image
// halton23_kernel
// sobol2_kernel
// zip_kernel
// stratify2_kernel
// init_image_coords
// colorspace_convert_image_half
// colorspace_convert_image_float
__global__ void shade_kernel_image(
    Vector2i resolution, 
    const Vector2f* __restrict__ positions, 
    const Array3f* __restrict__ colors,
    Array4f* __restrict__ frame_buffer,
    float* __restrict__ depth_buffer,
    bool linear_colors
) {
    // GET FRAME_BUFFER AND DEPTH_BUFFER CONTENT
}
// eval_image_kernel_and_snap
// train_image

void Testbed::render_image(CudaRenderBuffer& render_buffer, hipStream_t stream) {
    auto res = render_buffer.in_resolution();

    size_t n_pixels = (size_t)res.x() * res.y();
    // uint32_t n_elements
    // m_image.render_coords.enlarge();
    // m_image.render_out.enlarge();

    // Generate 2D coords at which to query the network
    // const dim3 threads = {16, 8, 1};
    // const dim3 blocks = { div_round_up()}
    // init_image_coords()
    if (m_image.type == EDataType::Float) {
        // linear_kernel
    }

    if (!m_render_ground_truth) {

    } else {
        // Render Ground Truth
        // GPUMatrix<float> positions_matrix((float*)m_image.render_coords.data(), 2, n_elements);\
        // colors_matrix
        // m_network->inference(stream, positions_matrix, colors_matrix);
    }

    // Splat colors to render texture
    // shader_kernel_image

}

void Testbed::load_image() {
    if (equals_case_insensitive(m_data_path.extension(), "exr")) {
        // LOAD EXR
    } else if (equals_case_insensitive(m_data_path.extension(), "bin")) {
        // LOAD binary
    } else {
        load_stbi_image();
    }

#ifdef COLOR_SPACE_CONVERT

#endif

}

void Testbed::load_stbi_image() {
    if (!m_data_path.exists()) {
        throw std::runtime_error{m_data_path.str() + " does not exist."};
    }

    // LOG

    // First: load an image that we'd liek to learn
    GPUMemory<float> image = load_stbi(m_data_path.str(), m_image.resolution.x(), m_image.resolution.y());
    m_image.data.resize(image.size() * sizeof(float));
    CUDA_CHECK_THROW(cudaMemory(m_image.data.data(), image.data(), image.size() * sizeof(float), cudaMemecpyDeviceToDevice));
    m_image.type = EDataType::Float;
}

NGP_NAMESPACE_END